#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>

#include "utils.h"

#define MAT_MUL_KERNEL_TILE_WIDTH 16

__global__ void matMulKernel(float* A, float* B, float* C, size_t i, size_t j, size_t k)
{

}

void matMul(float* A, float* B, float* C, size_t i, size_t j, size_t k)
{
    float *A_d = nullptr;
    float *B_d = nullptr;
    float *C_d = nullptr;

    checkCudaErrors(hipMalloc(&A_d, i * j * sizeof(float)));
    checkCudaErrors(hipMalloc(&B_d, j * k * sizeof(float)));
    checkCudaErrors(hipMalloc(&C_d, i * k * sizeof(float)));

    checkCudaErrors(hipMemcpy(A_d, A, i * j * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B_d, B, j * k * sizeof(float), hipMemcpyHostToDevice));

    dim3 block_dim { MAT_MUL_KERNEL_TILE_WIDTH, MAT_MUL_KERNEL_TILE_WIDTH ,1};

    matMulKernel<<<128, block_dim>>>(A_d, B_d, C_d, i, j, k);

    checkCudaErrors(hipMemcpy(C, C_d, i * k * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));
}