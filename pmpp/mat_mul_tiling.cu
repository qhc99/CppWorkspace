#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>

#include "utils.h"

#define MAT_MUL_KERNEL_TILE_WIDTH 16

__global__ void matMulTilingKernel(float* A, float* B, float* C, int i, int j, int k)
{

}

void matMulTiling(float* A, float* B, float* C, int i, int j, int k)
{
    float *A_d, *B_d, *C_d;

    checkCudaErrors(hipMalloc((void**)&A_d, i * j * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&B_d, j * k * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&C_d, i * k * sizeof(float)));

    checkCudaErrors(hipMemcpy(A_d, A, i * j * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B_d, B, j * k * sizeof(float), hipMemcpyHostToDevice));

    dim3 block_dim { MAT_MUL_KERNEL_TILE_WIDTH, MAT_MUL_KERNEL_TILE_WIDTH ,1};

    matMulTilingKernel<<<128, block_dim>>>(A_d, B_d, C_d, i, j, k);

    checkCudaErrors(hipMemcpy(C, C_d, i * k * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));
}