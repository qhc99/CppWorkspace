
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    float *A_d, *B_d,* C_d;
    int size = n * sizeof(float);
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n / 256.), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(B_d, B, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}