#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void vecAddKernel(float* A, float* B, float* C, size_t n)
{
    size_t i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, size_t n)
{
    float *A_d = nullptr;
    float *B_d = nullptr;
    float *C_d = nullptr;
    size_t size = n * sizeof(float);

    checkCudaError(hipMalloc(&A_d, size));
    checkCudaError(hipMalloc(&B_d, size));
    checkCudaError(hipMalloc(&C_d, size));

    checkCudaError(hipMemcpy(A_d, A, size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(B_d, B, size, hipMemcpyHostToDevice));

    vecAddKernel<<<static_cast<unsigned int>(ceil(static_cast<double>(n) / 256.)), 256>>>(A_d, B_d, C_d, n);
    checkCudaLastError();
    checkCudaError(hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost));

    checkCudaError(hipFree(A_d));
    checkCudaError(hipFree(B_d));
    checkCudaError(hipFree(C_d));
}