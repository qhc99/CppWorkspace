#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>

#include "utils.h"

__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n)
{
    float *A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    checkCudaErrors(hipMalloc((void**)&A_d, size));
    checkCudaErrors(hipMalloc((void**)&B_d, size));
    checkCudaErrors(hipMalloc((void**)&C_d, size));

    checkCudaErrors(hipMemcpy(A_d, A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B_d, B, size, hipMemcpyHostToDevice));

    // Cannot handle large n
    vecAddKernel<<<static_cast<unsigned int>(ceil(n / 256.)), 256>>>(A_d, B_d, C_d, n);

    checkCudaErrors(hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));
}