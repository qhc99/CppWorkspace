#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>

#include "utils.h"

__global__ void vecAddKernel(float* A, float* B, float* C, size_t n)
{
    size_t i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, size_t n)
{
    float *A_d = nullptr;
    float *B_d = nullptr;
    float *C_d = nullptr;
    size_t size = n * sizeof(float);

    checkCudaErrors(hipMalloc(&A_d, size));
    checkCudaErrors(hipMalloc(&B_d, size));
    checkCudaErrors(hipMalloc(&C_d, size));

    checkCudaErrors(hipMemcpy(A_d, A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B_d, B, size, hipMemcpyHostToDevice));

    // Cannot handle large n
    vecAddKernel<<<static_cast<unsigned int>(ceil(static_cast<double>(n) / 256.)), 256>>>(A_d, B_d, C_d, n);

    checkCudaErrors(hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(A_d));
    checkCudaErrors(hipFree(B_d));
    checkCudaErrors(hipFree(C_d));
}